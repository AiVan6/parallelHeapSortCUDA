#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>
#include <string>
#include <hip/hip_runtime.h>

/* ������������� ���������� �� CUDA */

void print(int arr[], int size) {
	for (int i = 0; i < size; i++) {
		std::cout << arr[i] << " ";
	}
	std::cout << "\n";
}

__device__ void swap(int &a, int &b) {
	int temp = a;
	a = b;
	b = temp;
}

__device__ void heapify(int arr[], int n, int root)
{
	int largest = root;
	int l = 2 * root + 1;
	int r = 2 * root + 2;

	if (l < n && arr[l] > arr[largest])
		largest = l;

	if (r < n && arr[r] > arr[largest])
		largest = r;

	if (largest != root)
	{
		//std::swap(arr[root], arr[largest]);
		swap(arr[root], arr[largest]);

		heapify(arr, n, largest);
	}
}

__global__ void sort(int arr[], int n)
{
	for (int i = n / 2 - 1; i >= 0; i--)
		heapify(arr, n, i);

	for (int i = n - 1; i >= 0; i--)
	{
		swap(arr[0], arr[i]);
		heapify(arr, i, 0);
	}
}

void fillArray(int arr[], int size)
{
	for (int i = 0; i < size; i++)
	{
		arr[i] = rand() % size;
	}
}

int main()
{
	long long int arrSize = 0;
	int N = 0;
	std::cout << "Enter array size: ";
	std::cin >> arrSize;
	std::cout << "Enter numbers of threads:";
	std::cin >> N;

	int* arr = new int[arrSize];
	fillArray(arr, arrSize);

	int* dev_arr;
	hipMalloc((void**)&dev_arr, arrSize * sizeof(int));
	hipMemcpy(dev_arr, arr, arrSize * sizeof(int), hipMemcpyHostToDevice);

	auto start = std::chrono::high_resolution_clock::now();

	sort << <1, N >> > (dev_arr, arrSize);
	
	auto end = std::chrono::high_resolution_clock::now();
	std::chrono::duration<float> duration = end - start;

	hipMemcpy(arr, dev_arr, arrSize * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(dev_arr);
	
	std::cout << "Parallel Time in nano: " << duration.count() << std::endl;


	
	delete[] arr;

	return 0;
}
